#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
<<<<<<< HEAD
<<<<<<< HEAD
    binary[i] = (x[i] >= 0) ? 1 : -1;
=======
    binary[i] = (x[i] > 0) ? 1 : -1;
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
    binary[i] = (x[i] > 0) ? 1 : -1;
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += abs(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


<<<<<<< HEAD
<<<<<<< HEAD
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
=======
__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
<<<<<<< HEAD
<<<<<<< HEAD
        mean += abs(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

void forward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        mean += abs(filters[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (filters[f*size + i] > 0) ? mean : -mean;
    }
}

void binarize_filters_gpu(float *filters, int n, int size, float *binary)
{
    binarize_filters_kernel<<<cuda_gridsize(n), BLOCK>>>(filters, n, size, binary);
    check_error(hipPeekAtLastError());
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_filters_gpu(l.filters_gpu, l.n, l.c*l.size*l.size, l.binary_filters_gpu);
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        swap_binary(&l);
    }

    if(l.xnor){
<<<<<<< HEAD
<<<<<<< HEAD
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(net.input_gpu, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        net.input_gpu = l.binary_input_gpu;
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        binarize_filters_gpu(l.filters_gpu, l.n, l.c*l.size*l.size, l.binary_filters_gpu);
        swap_binary(&l);
        binarize_gpu(state.input, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        state.input = l.binary_input_gpu;
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
    }

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
<<<<<<< HEAD
<<<<<<< HEAD
                net.input_gpu,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                net.workspace,
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
                state.input,
                l.filterDesc,
                l.filters_gpu,
                l.convDesc,
                l.fw_algo,
                state.workspace,
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
<<<<<<< HEAD
<<<<<<< HEAD
        im2col_ongpu(net.input_gpu + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, net.workspace);
        float * a = l.weights_gpu;
        float * b = net.workspace;
=======
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        float * a = l.filters_gpu;
        float * b = state.workspace;
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        float * a = l.filters_gpu;
        float * b = state.workspace;
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
#endif

    if (l.batch_normalize) {
<<<<<<< HEAD
<<<<<<< HEAD
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
=======
        forward_batchnorm_layer_gpu(l, state);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
        forward_batchnorm_layer_gpu(l, state);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592

    activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
}

<<<<<<< HEAD
<<<<<<< HEAD
__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size/2.);
    int h_offset = -(size/2.);

    int out_index = j + w*(i + h*(k + c*b));
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w*(cur_h + h*(k + b*c));
            int valid = (cur_h >= 0 && cur_h < h &&
                    cur_w >= 0 && cur_w < w);
            delta[out_index] += valid ? rate*(x[index] - x[out_index]) : 0;
        }
    }
}

extern "C" void smooth_layer(layer l, int size, float rate)
{
    int h = l.out_h;
    int w = l.out_w;
    int c = l.out_c;

    size_t n = h*w*c*l.batch;

    smooth_kernel<<<cuda_gridsize(n), BLOCK>>>(l.output_gpu, n, l.w, l.h, l.c, size, rate, l.delta_gpu);
    check_error(hipPeekAtLastError());
}

void backward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    if(l.smooth){
        smooth_layer(l, 5, l.smooth);
    }
    constrain_ongpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = net.input_gpu;

    if(l.xnor) net.input_gpu = l.binary_input_gpu;
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
    }
    float *original_input = state.input;

    if(l.xnor) state.input = l.binary_input_gpu;
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
<<<<<<< HEAD
<<<<<<< HEAD
            net.input_gpu,
=======
            state.input,
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
            state.input,
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
<<<<<<< HEAD
<<<<<<< HEAD
            net.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(net.delta_gpu){
        if(l.binary || l.xnor) swap_binary(&l);
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
            state.workspace,
            l.workspace_size,
            &one,
            l.dfilterDesc,
            l.filter_updates_gpu);

    if(state.delta){
        if(l.binary || l.xnor) swap_binary(&l);
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.filterDesc,
                l.filters_gpu,
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
<<<<<<< HEAD
<<<<<<< HEAD
                net.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                net.delta_gpu);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
                state.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                state.delta);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
    }

#else
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = l.out_w*l.out_h;

    int i;
    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
<<<<<<< HEAD
<<<<<<< HEAD
        float * b = net.workspace;
        float * c = l.weight_updates_gpu;

        im2col_ongpu(net.input_gpu + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, net.workspace);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(net.delta_gpu){
            if(l.binary || l.xnor) swap_binary(&l);
            float * a = l.weights_gpu;
            float * b = l.delta_gpu;
            float * c = net.workspace;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(net.workspace, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, net.delta_gpu + i*l.c*l.h*l.w);
            if(l.binary || l.xnor) {
                swap_binary(&l);
            }
            if(l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        float * b = state.workspace;
        float * c = l.filter_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            if(l.binary || l.xnor) swap_binary(&l);
            float * a = l.filters_gpu;
            float * b = l.delta_gpu;
            float * c = state.workspace;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(state.workspace, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            if(l.binary || l.xnor) {
                swap_binary(&l);
            }
            if(l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, state.delta + i*l.c*l.h*l.w);
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        }
    }
#endif
}

void pull_convolutional_layer(convolutional_layer layer)
{
<<<<<<< HEAD
<<<<<<< HEAD
    cuda_pull_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
=======
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
<<<<<<< HEAD
<<<<<<< HEAD
    if (layer.adam){
        cuda_pull_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_pull_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
=======
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
}

void push_convolutional_layer(convolutional_layer layer)
{
<<<<<<< HEAD
<<<<<<< HEAD
    cuda_push_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
=======
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
<<<<<<< HEAD
<<<<<<< HEAD
    if (layer.adam){
        cuda_push_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_push_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void adam_update_gpu(float *w, float *d, float *m, float *v, float B1, float B2, float eps, float decay, float rate, int n, int batch, int t)
{
    scal_ongpu(n, B1, m, 1);
    scal_ongpu(n, B2, v, 1);
    axpy_ongpu(n, -decay*batch, w, 1, d, 1);

    axpy_ongpu(n, (1-B1), d, 1, m, 1);
    mul_ongpu(n, d, 1, d, 1);
    axpy_ongpu(n, (1-B2), d, 1, v, 1);

    adam_gpu(n, w, m, v, B1, B2, rate/batch, eps, t);
    fill_ongpu(n, 0, d, 1);
}

void update_convolutional_layer_gpu(layer l, int batch, float learning_rate, float momentum, float decay)
{
    int size = l.size*l.size*l.c*l.n;

    if(l.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, size, batch, l.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
        }
    }else{
        axpy_ongpu(size, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_ongpu(size, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_ongpu(size, momentum, l.weight_updates_gpu, 1);

        axpy_ongpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_ongpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_ongpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_ongpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
}

void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);

    axpy_ongpu(size, -decay*batch, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, learning_rate/batch, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, momentum, layer.filter_updates_gpu, 1);
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
}


