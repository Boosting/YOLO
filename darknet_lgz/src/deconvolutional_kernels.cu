#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
<<<<<<< HEAD
<<<<<<< HEAD
#include "batchnorm_layer.h"
=======
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

<<<<<<< HEAD
<<<<<<< HEAD
extern "C" void forward_deconvolutional_layer_gpu(layer l, network net)
{
    int i;

    int m = l.size*l.size*l.n;
    int n = l.h*l.w;
    int k = l.c;

    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

    for(i = 0; i < l.batch; ++i){
        float *a = l.weights_gpu;
        float *b = net.input_gpu + i*l.c*l.h*l.w;
        float *c = net.workspace;

        gemm_ongpu(1,0,m,n,k,1,a,m,b,n,0,c,n);

        col2im_ongpu(net.workspace, l.out_c, l.out_h, l.out_w, l.size, l.stride, l.pad, l.output_gpu+i*l.outputs);
    }
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    activate_array_ongpu(l.output_gpu, l.batch*l.n*l.out_w*l.out_h, l.activation);
}

extern "C" void backward_deconvolutional_layer_gpu(layer l, network net)
{
    int i;

    constrain_ongpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    //if(net.delta_gpu) memset(net.delta_gpu, 0, l.batch*l.h*l.w*l.c*sizeof(float));

    for(i = 0; i < l.batch; ++i){
        int m = l.c;
        int n = l.size*l.size*l.n;
        int k = l.h*l.w;

        float *a = net.input_gpu + i*m*k;
        float *b = net.workspace;
        float *c = l.weight_updates_gpu;

        im2col_ongpu(l.delta_gpu + i*l.outputs, l.out_c, l.out_h, l.out_w, 
                l.size, l.stride, l.pad, b);
        gemm_ongpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

        if(net.delta_gpu){
            int m = l.c;
            int n = l.h*l.w;
            int k = l.size*l.size*l.n;

            float *a = l.weights_gpu;
            float *b = net.workspace;
            float *c = net.delta_gpu + i*n*m;

            gemm_ongpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
extern "C" void forward_deconvolutional_layer_gpu(deconvolutional_layer layer, network_state state)
{
    int i;
    int out_h = deconvolutional_out_height(layer);
    int out_w = deconvolutional_out_width(layer);
    int size = out_h*out_w;

    int m = layer.size*layer.size*layer.n;
    int n = layer.h*layer.w;
    int k = layer.c;

    fill_ongpu(layer.outputs*layer.batch, 0, layer.output_gpu, 1);

    for(i = 0; i < layer.batch; ++i){
        float *a = layer.filters_gpu;
        float *b = state.input + i*layer.c*layer.h*layer.w;
        float *c = layer.col_image_gpu;

        gemm_ongpu(1,0,m,n,k,1,a,m,b,n,0,c,n);

        col2im_ongpu(c, layer.n, out_h, out_w, layer.size, layer.stride, 0, layer.output_gpu+i*layer.n*size);
    }
    add_bias_gpu(layer.output_gpu, layer.biases_gpu, layer.batch, layer.n, size);
    activate_array(layer.output_gpu, layer.batch*layer.n*size, layer.activation);
}

extern "C" void backward_deconvolutional_layer_gpu(deconvolutional_layer layer, network_state state)
{
    float alpha = 1./layer.batch;
    int out_h = deconvolutional_out_height(layer);
    int out_w = deconvolutional_out_width(layer);
    int size = out_h*out_w;
    int i;

    gradient_array(layer.output_gpu, size*layer.n*layer.batch, layer.activation, layer.delta_gpu);
    backward_bias(layer.bias_updates_gpu, layer.delta, layer.batch, layer.n, size);

    if(state.delta) memset(state.delta, 0, layer.batch*layer.h*layer.w*layer.c*sizeof(float));

    for(i = 0; i < layer.batch; ++i){
        int m = layer.c;
        int n = layer.size*layer.size*layer.n;
        int k = layer.h*layer.w;

        float *a = state.input + i*m*n;
        float *b = layer.col_image_gpu;
        float *c = layer.filter_updates_gpu;

        im2col_ongpu(layer.delta_gpu + i*layer.n*size, layer.n, out_h, out_w, 
                layer.size, layer.stride, 0, b);
        gemm_ongpu(0,1,m,n,k,alpha,a,k,b,k,1,c,n);

        if(state.delta){
            int m = layer.c;
            int n = layer.h*layer.w;
            int k = layer.size*layer.size*layer.n;

            float *a = layer.filters_gpu;
            float *b = layer.col_image_gpu;
            float *c = state.delta + i*n*m;

            gemm(0,0,m,n,k,1,a,k,b,n,1,c,n);
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
        }
    }
}

<<<<<<< HEAD
<<<<<<< HEAD
extern "C" void pull_deconvolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

extern "C" void push_deconvolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_deconvolutional_layer_gpu(layer l, int batch, float learning_rate, float momentum, float decay)
{
    int size = l.size*l.size*l.c*l.n;

    if(l.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, size, batch, l.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
        }
    }else{
        axpy_ongpu(size, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_ongpu(size, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_ongpu(size, momentum, l.weight_updates_gpu, 1);

        axpy_ongpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_ongpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_ongpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_ongpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
=======
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
extern "C" void pull_deconvolutional_layer(deconvolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void push_deconvolutional_layer(deconvolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void update_deconvolutional_layer_gpu(deconvolutional_layer layer, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, learning_rate, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(size, -decay, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, learning_rate, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, momentum, layer.filter_updates_gpu, 1);
<<<<<<< HEAD
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
=======
>>>>>>> 07267f401b3d9c82c5f695f932c9f504d2b6a592
}

