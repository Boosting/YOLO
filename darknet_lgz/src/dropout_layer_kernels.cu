#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

<<<<<<< HEAD
void forward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if (!net.train) return;
=======
void forward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if (!state.train) return;
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */

<<<<<<< HEAD
    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if(!net.delta_gpu) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
=======
    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(state.input, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if(!state.delta) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(state.delta, size, layer.rand_gpu, layer.probability, layer.scale);
>>>>>>> b5b3d7367411302dd6e73c8fe583d6860a786445
    check_error(hipPeekAtLastError());
}
